
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>

#define Z 2
#define Y 5
#define X 5
#define xBound X / 2
#define yBound Y / 2
#define SCALE 8

int read_png(const char* filename, unsigned char** image, unsigned* height, 
             unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, 
               const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

__constant__ char mask[Z][Y][X] = { { { -1, -4, -6, -4, -1 },
                                        { -2, -8, -12, -8, -2 },
                                        { 0, 0, 0, 0, 0 },
                                        { 2, 8, 12, 8, 2 },
                                        { 1, 4, 6, 4, 1 } },
                                      { { -1, -2, 0, 2, 1 },
                                        { -4, -8, 0, 8, 4 },
                                        { -6, -12, 0, 12, 6 },
                                        { -4, -8, 0, 8, 4 },
                                        { -1, -2, 0, 2, 1 } } };

inline __device__ int bound_check(int val, int lower, int upper) {
    if (val >= lower && val < upper)
        return 1;
    else
        return 0;
}

__global__ void sobel(unsigned char *s, unsigned char *t, unsigned height, unsigned width, unsigned channels) {

    __shared__ unsigned char R_new[5][260];
    __shared__ unsigned char G_new[5][260];
    __shared__ unsigned char B_new[5][260];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    int x = tid;
    if (x >= width) return;

    int y = blockIdx.y;

    
    #pragma unroll 5
    for (int v = -yBound; v <= yBound; ++v) {
        if(!bound_check(y + v, 0, height)) return;
        
            R_new[v + yBound][threadIdx.x +xBound] = s[channels * (width * (y + v) + x) + 2];
            G_new[v + yBound][threadIdx.x +xBound] = s[channels * (width * (y + v) + x) + 1];
            B_new[v + yBound][threadIdx.x +xBound] = s[channels * (width * (y + v) + x) + 0];

            if(threadIdx.x == 0){
                if(x != 0){
                    #pragma unroll 2
                    for(int i=0; i<2; i++)
                    {
                        R_new[v + yBound][i] = s[channels * (width * (y + v) + x + i - 2) + 2];
                        G_new[v + yBound][i] = s[channels * (width * (y + v) + x + i - 2) + 1];
                        B_new[v + yBound][i] = s[channels * (width * (y + v) + x + i - 2) + 0];
                    }
                } 
            }
            else if(threadIdx.x == blockDim.x - 1){
                if(x + blockDim.x < width)
                {
                    #pragma unroll 2
                    for(int i=0; i<2; i++)
                    {
                        R_new[v + yBound][threadIdx.x + xBound + i + 1] = s[channels * (width * (y + v) + x + i + 1) + 2];
                        G_new[v + yBound][threadIdx.x + xBound + i + 1] = s[channels * (width * (y + v) + x + i + 1) + 1];
                        B_new[v + yBound][threadIdx.x + xBound + i + 1] = s[channels * (width * (y + v) + x + i + 1) + 0];
                    }   
                }
            }
        
    }

    __syncthreads();

    
        /* Z axis of mask */
        float val[Z][3];
        #pragma unroll 2
        for (int i = 0; i < Z; ++i) {

            val[i][2] = 0.;
            val[i][1] = 0.;
            val[i][0] = 0.;

            /* Y and X axis of mask */
            #pragma unroll 5
            for (int v = -yBound; v <= yBound; ++v) {
                #pragma unroll 5
                for (int u = -xBound; u <= xBound; ++u) {
                    if (bound_check(x + u, 0, width) && bound_check(y + v, 0, height)) {
                        const unsigned char R = R_new[v + yBound][threadIdx.x + u + xBound];
                        const unsigned char G = G_new[v + yBound][threadIdx.x + u + xBound];
                        const unsigned char B = B_new[v + yBound][threadIdx.x + u + xBound];
                        val[i][2] += R * mask[i][u + xBound][v + yBound];
                        val[i][1] += G * mask[i][u + xBound][v + yBound];
                        val[i][0] += B * mask[i][u + xBound][v + yBound];
                    }
                }
            }
        }
        float totalR = 0.;
        float totalG = 0.;
        float totalB = 0.;
        //#pragma unroll 2
        for (int i = 0; i < Z; ++i) {
            totalR += val[i][2] * val[i][2];
            totalG += val[i][1] * val[i][1];
            totalB += val[i][0] * val[i][0];
        }
        totalR = sqrt(totalR) / SCALE;
        totalG = sqrt(totalG) / SCALE;
        totalB = sqrt(totalB) / SCALE;
        const unsigned char cR = (totalR > 255.) ? 255 : totalR;
        const unsigned char cG = (totalG > 255.) ? 255 : totalG;
        const unsigned char cB = (totalB > 255.) ? 255 : totalB;
        t[channels * (width * y + x) + 2] = cR;
        t[channels * (width * y + x) + 1] = cG;
        t[channels * (width * y + x) + 0] = cB;
    
}

int main(int argc, char **argv) {
    assert(argc == 3);
    unsigned height, width, channels;
    unsigned char *src = NULL, *dst;
    unsigned char *dsrc, *ddst;

    /* read the image to src, and get height, width, channels */
    if (read_png(argv[1], &src, &height, &width, &channels)) {
        std::cerr << "Error in read png" << std::endl;
        return -1;
    }

    dst = (unsigned char *)malloc(height * width * channels * sizeof(unsigned char));
    hipHostRegister(src, height * width * channels * sizeof(unsigned char), hipHostRegisterDefault);

    // cudaMalloc(...) for device src and device dst
    hipMalloc(&dsrc, height * width * channels * sizeof(unsigned char));
    hipMalloc(&ddst, height * width * channels * sizeof(unsigned char));

    // cudaMemcpy(...) copy source image to device (mask matrix if necessary)
    hipMemcpy(dsrc, src, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // decide to use how many blocks and threads
    const int num_threads = 256;
    //const int num_blocks = height / num_threads + 1;
    dim3 num_blocks(int(width / num_threads + 1), height);

    // launch cuda kernel
    sobel<<<num_blocks, num_threads>>>(dsrc, ddst, height, width, channels);

    // cudaMemcpy(...) copy result image to host
    hipMemcpy(dst, ddst, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    write_png(argv[2], dst, height, width, channels);
    free(src);
    free(dst);
    hipFree(dsrc);
    hipFree(ddst);
    return 0;
}
